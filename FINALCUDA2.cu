#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <stdlib.h>
#include <math.h>
#include <string.h>

__host__ __device__ void swap(char *x, char *y)
{
	char temp;
	temp = *x;
	*x = *y;
	*y = temp;
}

__device__ long hash(char *str)
{
	unsigned long hash = 5381;
	int c;

	while (c = *str++)
		hash = ((hash << 5) + hash) + c; 

	return hash;

}

__device__ void permute_device(char *a, int i, int n, int tid, int* count,char *d_b,int *d)
{
	if (i == n) {
		char* c = a - 1;
		//b[0] = c[0]; b[1] = c[1]; b[2] = c[2]; b[3] = c[3];
		//printf("Permutation no %i from thread no %i is equal to %s\n", count[0], tid, c); count[0] = count[0] + 1;
		long h1 = hash(c);
		long h2 = hash(d_b);

		if (h1 == h2)
		{
			printf("\nHashValue of found string=%d\n", h2);
			d[0] = 1;
			
			//return;
		}
		
	}
	else
	{
		for (int j = i; j <= n; j++)
		{
			swap((a + i), (a + j));
			permute_device(a, i + 1, n, tid, count,d_b,d);
			swap((a + i), (a + j));
		}
	}
}


__global__ void q4(char *a, char *b, int *C, int *d,int len)
{
	int id = threadIdx.x;
	int s = C[id];
	int c = C[id + 1];

	int count;
	
	char buf[20];
	int j = 0;
	for (int i = s + 1;i < c;i++) {
		buf[j++] = a[i];
	}
	
		long h1 = hash(buf);
		long h2 = hash(b);

		if (h1 == h2)
		{
			printf("\nHashValue of found string=%d\n", h2);
			d[0] = 1;
			//return;
		}


}

__global__ void permute_kernel(char* d_A, int size,char* d_B,int *d) {


	int tid = threadIdx.x;
	int count[1]; count[0] = 0;

	char local_array[10];

	for (int i = 0; i < size; i++) {
		local_array[i] = d_A[i];
	}
	if (threadIdx.x <= size-1) {
		swap(local_array + threadIdx.x, local_array);
		permute_device(local_array + 1, 0, size-1, tid, count,d_B,d);
	}

}

int main(int argc, char* argv[])
{
	clock_t start_t = clock();
	char h_b[30]="operco";
	int d = 0;
	FILE *fp;
	long lSize;
	char *buffer;

	fp = fopen("test.txt", "rb");
	if (!fp) perror("test.txt"), exit(1);

	fseek(fp, 0L, SEEK_END);
	lSize = ftell(fp);
	rewind(fp);

	/* allocate memory for entire content */
	buffer = (char*)calloc(1, lSize + 1);
	if (!buffer) fclose(fp), fputs("memory alloc fails", stderr), exit(1);

	/* copy the file into the buffer */
	if (1 != fread(buffer, lSize, 1, fp))
		fclose(fp), free(buffer), fputs("entire read fails", stderr), exit(1);

	/* do your work here, buffer is a string contains the whole text */
	//printf("%s", buffer);

	fclose(fp);

	
	//printf("Enter the password \n");
	//scanf("%s",h_b);
	int spaces[20000];
	int j = 0;
	spaces[j++] = -1;
	for (int i = 0;buffer[i] != '\0';i++) {
		
		if (buffer[i] == ' ')
			
			spaces[j++] = i;
	}
	printf("SPACES= %d %d %d %d", spaces[0],spaces[1],spaces[2],spaces[3]);
	//int k=strlen(spaces);
	int k = j;
	int n = strlen(buffer);
	int m = strlen(h_b);
	int dd[1];
	dd[0] = 0;






	// Device input vectors
	char *d_a;
	char *d_b;
	int *d_c;
	int *d_d;

	size_t bytes = n * sizeof(char);

	// Allocate memory for each vector on GPU
	hipMalloc(&d_a, n * sizeof(char));
	hipMalloc(&d_b, 30 * sizeof(char));
	hipMalloc(&d_c, k * sizeof(int));
	hipMalloc(&d_d, 1* sizeof(int));




	//int i;



	// Copy host vectors to device
	hipMemcpy(d_a, buffer, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, m * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(d_c, spaces, k * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_d, dd, 1 * sizeof(int), hipMemcpyHostToDevice);

	q4 << <1, j >> > (d_a, d_b, d_c, d_d,m);


	hipMemcpy(dd, d_d, 1* sizeof(int), hipMemcpyDeviceToHost);
	printf("\n");
	//int sum = 0;
	printf("Searching in the Dictionary \n");
		if (dd[0] == 1) {
			printf("Found %s in the Dictionary And Hash of password  has been calculated\n",h_b);
			//return 1;
		}
		else if (dd[0] == 0)
		{
			printf("Unable to find Password in Dictionary\n");
			printf("Finding Permutations of the words only if max length of string is 7\n");
			int t = 0;
			if(m<=7){
				for (int r = 0;r < k - 1;r++)
				{
					char buf[50];
					char* d_k;
					int s = spaces[r];
					int c = spaces[r + 1];
					//printf("m=%d\n", m);
					//printf("c-s-1=%d\n", c-s-1);

					if(m != (c - s - 1))
					{
						continue;
					}
					for (int i = s + 1;i < c;i++) {
						buf[t++] = buffer[i];
					}

					hipMalloc((void**)&d_k, sizeof(buf));
					hipMemcpy(d_k, buf, sizeof(buf), hipMemcpyHostToDevice);

					permute_kernel << <1, t >> > (d_k, t, d_b, d_d);
					hipMemcpy(dd, d_d, 1 * sizeof(int), hipMemcpyDeviceToHost);
					if (dd[0] == 1) {
						printf("Found string '%s' while calculating permutation\n", h_b);
						//return 1;
						break;
					}
					memset(buf, 0, sizeof(buf));
					t = 0;

				}

			}
		}
		if (dd[0] == 0) {
			printf("Unable to find string even after computing permutation\n", h_b);
		}


		clock_t end_t = clock();
		clock_t total_t = (end_t - start_t);
		printf("Elapsed Time:%.3f seconds\n", (double)total_t / ((double)CLOCKS_PER_SEC));
	// Release device memory
	
		hipFree(d_a);
		hipFree(d_b);
		hipFree(d_c);
		hipFree(d_d);
		//hipFree(d_k);


		//free(h_b);
	

	return 0;
}



