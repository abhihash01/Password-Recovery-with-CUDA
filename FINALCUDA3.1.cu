#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <stdlib.h>
#include <math.h>
#include <string.h>

__host__ __device__ void swap(char *x, char *y)
{
	char temp;
	temp = *x;
	*x = *y;
	*y = temp;
}

__device__ long hash(char *str)
{
	unsigned long hash = 5381;
	int c;

	while (c = *str++)
		hash = ((hash << 5) + hash) + c; 

	return hash;

}

__device__ void permute_device(char *a, int i, int n, int tid, int* count,long d_b,int *d)
{
	if (i == n) {
		char* c = a - 1;
		printf("Permutation no %i from thread no %i is equal to %s\n", count[0], tid, c); count[0] = count[0] + 1;
		long h1 = hash(c);
		//long h2 = hash(d_b);
		//printf("\nHashValue=%d\n", h1);
		
		if (h1 == d_b)
		{
			printf("\nHashValue of found string=%d\n", h1);
			printf("\n Password is '%s'\n", c);
			d[0] = 1;
		}
		
		
	}
	else
	{
		for (int j = i; j <= n; j++)
		{
			swap((a + i), (a + j));
			permute_device(a, i + 1, n, tid, count,d_b,d);
			swap((a + i), (a + j));
		}
	}
}


__global__ void q4(char *a, long b, int *C, int *d)
{
	int id = threadIdx.x;
	int s = C[id];
	int c = C[id + 1];

	int count;
	
	char buf[20];
	int j = 0;
	for (int i = s + 1;i < c;i++) {
		buf[j++] = a[i];
	}
	
		long h1 = hash(buf);
		

		if (h1 == b)
		{
			printf("\nHashValue of found string=%d\n", h1);
			printf("\n Password is '%s'\n", buf);
			d[0] = 1;
			//return;
		}


}

__global__ void permute_kernel(char* d_A, int size,long d_B,int *d) {

	
	int tid = threadIdx.x;
	int count[1]; count[0] = 0;

	char local_array[20];
	int i;
	for ( i = 0; i < size; i++) {
		local_array[i] = d_A[i];
	}
	local_array[i] = '\0';
	//printf("string=%s\n", local_array);
	if (threadIdx.x <= size-1) {
		swap(local_array + threadIdx.x, local_array);
		permute_device(local_array + 1, 0, size-1, tid, count,d_B,d);
	}

}

int main(int argc, char* argv[])
{
	clock_t start_t = clock();
	long h_b=276005754;
	int d = 0;
	FILE *fp;
	long lSize;
	char *buffer;

	fp = fopen("test.txt", "rb");
	if (!fp) perror("test.txt"), exit(1);

	fseek(fp, 0L, SEEK_END);
	lSize = ftell(fp);
	rewind(fp);

	/* allocate memory for entire content */
	buffer = (char*)calloc(1, lSize + 1);
	if (!buffer) fclose(fp), fputs("memory alloc fails", stderr), exit(1);

	/* copy the file into the buffer */
	if (1 != fread(buffer, lSize, 1, fp))
		fclose(fp), free(buffer), fputs("entire read fails", stderr), exit(1);

	/* do your work here, buffer is a string contains the whole text */
	//printf("%s", buffer);

	fclose(fp);

	
	//printf("Enter the password \n");
	//scanf("%s",h_b);
	int spaces[20000];
	int j = 0;
	spaces[j++] = -1;
	for (int i = 0;buffer[i] != '\0';i++) {
		
		if (buffer[i] == ' ')
			
			spaces[j++] = i;
	}
	printf("SPACES= %d %d %d %d %d %d\n", spaces[0],spaces[1],spaces[2],spaces[3], spaces[4], spaces[5]);
	//int k=strlen(spaces);
	int k = j;
	int n = strlen(buffer);
	//int m = strlen(h_b);
	int dd[1];
	dd[0] = 0;






	// Device input vectors
	char *d_a;
	char *d_b;
	int *d_c;
	int *d_d;

	size_t bytes = n * sizeof(char);

	// Allocate memory for each vector on GPU
	hipMalloc(&d_a, n * sizeof(char));
	//hipMalloc(&d_b, 30 * sizeof(int));
	hipMalloc(&d_c, k * sizeof(int));
	hipMalloc(&d_d, 1* sizeof(int));




	//int i;



	// Copy host vectors to device
	hipMemcpy(d_a, buffer, bytes, hipMemcpyHostToDevice);
	//hipMemcpy(d_b, h_b, 10 * sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(d_c, spaces, k * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_d, dd, 1 * sizeof(int), hipMemcpyHostToDevice);
	//int m = 5;
	printf("Searching in the Dictionary \n");
	q4 << <1, j >> > (d_a, h_b, d_c, d_d);


	hipMemcpy(dd, d_d, 1* sizeof(int), hipMemcpyDeviceToHost);
	printf("\n");
	//int sum = 0;

		if (dd[0] == 1) {
			printf("Found String in the Dictionary And Hash of password  has been calculated\n");
			//return 1;
		}
		else if (dd[0] == 0)
		{
			printf("Unable to find Password in Dictionary\n");
			printf("Finding Permutations of the words only if max length of string is 7\n");
			int t = 0;
			
			char buf[50];
			char* d_k;
			
				for (int r = 0;r < k;r++)
				{
					memset(buf, 0, sizeof(buf));
					
					
					
					int s = spaces[r];
					int c = spaces[r + 1];
					//printf("m=%d\n", m);
					//printf("c-s-1=%d\n", c-s-1);

					if(7<=(c - s - 1))
					{
						continue;
					}
					
					for (int i = s + 1;i < c;i++) {
						buf[t++] = buffer[i];
					}
					printf("BuFFER=%s", buf);
					printf("t=%d", t);
					hipMalloc((void**)&d_k, sizeof(buf));
					//printf("ll\n");

					
					hipMemcpy(d_k, buf, sizeof(buf), hipMemcpyHostToDevice);

					permute_kernel << <1, t >> > (d_k, t, h_b, d_d);
					hipMemcpy(dd, d_d, 1 * sizeof(int), hipMemcpyDeviceToHost);
					if (dd[0] == 1) {
						printf("Found string while calculating permutation\n");
						//return 1;
						break;
					}
					//memset(buf, 0, sizeof(buf));
					t = 0;
					hipFree(d_k);
					
				}

			
		}
		if (dd[0] == 0) {
			printf("Unable to find string even after computing permutation\n", h_b);
		}


		clock_t end_t = clock();
		clock_t total_t = (end_t - start_t);
		printf("Elapsed Time:%.3f seconds\n", (double)total_t / ((double)CLOCKS_PER_SEC));
	// Release device memory
	
		hipFree(d_a);
		//hipFree(d_b);
		hipFree(d_c);
		hipFree(d_d);
		//hipFree(d_k);


		//free(h_b);
	

	return 0;
}



