#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <stdlib.h>
#include <math.h>
#include <string.h>

__host__ __device__ void swap(char *x, char *y)
{
	char temp;
	temp = *x;
	*x = *y;
	*y = temp;
}

__device__ long hash(char *str)
{
	unsigned long hash = 5381;
	int c;

	while (c = *str++)
		hash = ((hash << 5) + hash) + c; 

	return hash;

}

__device__ void permute(char *a, int i, int n, int tid, int* count,long d_b,int *d,int *e)
{
	if (i == n) {
		char* c = a - 1;
		//printf("Permutation no %i from thread no %i is equal to %s\n", count[0], tid, c); count[0] = count[0] + 1;
		long h1 = hash(c);
		//long h2 = hash(d_b);
		//printf("\nHashValue=%d\n", h1);
		
		if (h1 == d_b)
		{
			printf("\nHashValue of found string=%d\n", h1);
			printf("\n Password is '%s'\n", c);
			d[0] = 1;
			e[0] = 1;
		}
		
		
	}
	else
	{
		for (int j = i; j <= n; j++)
		{
			swap((a + i), (a + j));
			permute(a, i + 1, n, tid, count,d_b,d,e);
			swap((a + i), (a + j));
		}
	}
}


__global__ void q4(char *a, long b, int *C, int *d)
{
	int id = threadIdx.x;
	int s = C[id];
	int c = C[id + 1];

	int count;
	
	char buf[20];
	int j = 0;
	for (int i = s + 1;i < c;i++) {
		buf[j++] = a[i];
	}
	
		long h1 = hash(buf);
		

		if (h1 == b)
		{
			printf("\nHashValue of found string=%d\n", h1);
			printf("\n Password is '%s'\n", buf);
			d[0] = 1;
			//return;
		}


}

__global__ void main_permute(char* d_A, int size,long d_B,int *d,int *e) {

	
	int tid = threadIdx.x;
	int count[1]; count[0] = 0;

	char local_array[20];
	int i;
	for ( i = 0; i < size; i++) {
		local_array[i] = d_A[i];
	}
	local_array[i] = '\0';
	//printf("string=%s\n", local_array);
	if (e[0] == 0) {
		if (threadIdx.x <= size - 1) {
			swap(local_array + threadIdx.x, local_array);
			permute(local_array + 1, 0, size - 2, tid, count, d_B, d,e);
		}
	}

}

int main(int argc, char* argv[])
{
	//long h_b=269574929;
	long int h_b=0;
	int d = 0;
	FILE *fp;
	long lSize;
	char *buffer;
	

	fp = fopen("test.txt", "rb");
	if (!fp) perror("test.txt"), exit(1);

	fseek(fp, 0L, SEEK_END);
	lSize = ftell(fp);
	rewind(fp);

	/* allocate memory for entire content */
	buffer = (char*)calloc(1, lSize + 1);
	if (!buffer) fclose(fp), fputs("memory alloc fails", stderr), exit(1);

	/* copy the file into the buffer */
	if (1 != fread(buffer, lSize, 1, fp))
		fclose(fp), free(buffer), fputs("entire read fails", stderr), exit(1);

	/* do your work here, buffer is a string contains the whole text */
	//printf("%s", buffer);

	fclose(fp);

	
	//printf("Enter the password \n");
	//scanf("%s",h_b);
	printf("enter hash value of password: ");
	scanf("%ld",&h_b);

	int spaces[20000];
	int j = 0;
	spaces[j++] = -1;
	for (int i = 0;buffer[i] != '\0';i++) {
		
		if (buffer[i] == ' ')
			
			spaces[j++] = i;
	}
	clock_t start_t = clock();
	printf("SPACES= %d %d %d %d %d %d\n", spaces[0],spaces[1],spaces[2],spaces[3], spaces[4], spaces[5]);
	//int k=strlen(spaces);
	int k = j;
	int n = strlen(buffer);
	//int m = strlen(h_b);
	int dd[1];
	dd[0] = 0;






	// Device input vectors
	char *d_a;
	char *d_b;
	int *d_c;
	int *d_d;
	int *d_e;

	size_t bytes = n * sizeof(char);

	// Allocate memory for each vector on GPU
	hipMalloc(&d_a, n * sizeof(char));
	//hipMalloc(&d_b, 30 * sizeof(int));
	hipMalloc(&d_c, k * sizeof(int));
	hipMalloc(&d_d, 1* sizeof(int));
	hipMalloc(&d_e, 1 * sizeof(int));




	//int i;

	int de[1];
	de[0] = 0;

	// Copy host vectors to device
	hipMemcpy(d_a, buffer, bytes, hipMemcpyHostToDevice);
	//hipMemcpy(d_b, h_b, 10 * sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(d_c, spaces, k * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_d, dd, 1 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_e, de, 1 * sizeof(int), hipMemcpyHostToDevice);
	//int m = 5;
	printf("Searching in the Dictionary \n");
	q4 << <1, j >> > (d_a, h_b, d_c, d_d);


	hipMemcpy(dd, d_d, 1* sizeof(int), hipMemcpyDeviceToHost);
	printf("\n");
	//int sum = 0;

		if (dd[0] == 1) {
			printf("Found String in the Dictionary And Hash of password  has been calculated\n");
			//return 1;
		}
		else if (dd[0] == 0)
		{
			printf("Unable to find Password in Dictionary\n");
			printf("Finding Permutations of the words\n");
			int t = 0;
			
			char buf[50];
			char* d_k;
			
				for (int r = 0;r < k;r++)
				{
					memset(buf, 0, sizeof(buf));
					
					
					
					int s = spaces[r];
					int c = spaces[r + 1];
					//printf("m=%d\n", m);
					//printf("c-s-1=%d\n", c-s-1);

					if(10<=(c - s - 1))
					{
						continue;
					}

					
					for (int i = s + 1;i < c;i++) {
						buf[t++] = buffer[i];
					}
					//printf("BuFFER=%s", buf);
					//printf("t=%d", t);
					hipMalloc((void**)&d_k, sizeof(buf));
					//printf("ll\n");

					
					hipMemcpy(d_k, buf, sizeof(buf), hipMemcpyHostToDevice);

					main_permute << <1, t >> > (d_k, t, h_b, d_d,d_e);
					hipMemcpy(dd, d_d, 1 * sizeof(int), hipMemcpyDeviceToHost);
					hipMemcpy(de, d_e, 1 * sizeof(int), hipMemcpyDeviceToHost);
					//hipMemcpy(de, d_e, 1 * sizeof(int), hipMemcpyDeviceToHost);
					if (dd[0] == 1) {
						printf("Found string while calculating permutation\n");
						//return 1;
						break;
					}
					//memset(buf, 0, sizeof(buf));
					t = 0;
					hipFree(d_k);
					
				}

			
		}
		if (dd[0] == 0) {
			printf("Unable to find string even after computing permutation\n", h_b);
		}


		clock_t end_t = clock();
		clock_t total_t = (end_t - start_t);
		printf("Elapsed Time:%.3f seconds\n", (double)total_t / ((double)CLOCKS_PER_SEC));
	// Release device memory
	
		hipFree(d_a);
		//hipFree(d_b);
		hipFree(d_c);
		hipFree(d_d);
		//hipFree(d_k);


		//free(h_b);
	

	return 0;
}



